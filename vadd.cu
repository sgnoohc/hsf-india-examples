
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <chrono>
using namespace std::chrono;

__global__ void vec_add(const float* A, const float* B, float* C, unsigned long long int N_data, unsigned long long int N_ops)
{

    unsigned long long int i_data = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_data < N_data)
    {
        for (unsigned i = 0; i < N_ops; ++i)
        {
            C[i_data] = A[i_data] + B[i_data];
        }
    }
}

int main(int argc, char** argv)
{

    std::cout << "#################################" << std::endl;
    std::cout << "#                               #" << std::endl;
    std::cout << "#                               #" << std::endl;
    std::cout << "#    Vector Addition Program    #" << std::endl;
    std::cout << "#            (GPU)              #" << std::endl;
    std::cout << "#                               #" << std::endl;
    std::cout << "#################################" << std::endl;

    unsigned long long int N_data = 10000000;
    unsigned long long int N_ops = 10000;

    auto start = high_resolution_clock::now();

    float* A_host = new float[N_data];
    float* B_host = new float[N_data];
    float* C_host = new float[N_data];

    for (unsigned int i = 0; i < N_data; ++i)
    {
        A_host[i] = i;
        B_host[i] = i * pow(-1, i);
    }

    auto mid1 = high_resolution_clock::now();

    // allocate memory on device
    float* A_device;
    float* B_device;
    float* C_device;
    hipMalloc((void**) &A_device, N_data * sizeof(float));
    hipMalloc((void**) &B_device, N_data * sizeof(float));
    hipMalloc((void**) &C_device, N_data * sizeof(float));

    auto mid2 = high_resolution_clock::now();

    // copy the host input data to the device input data
    hipMemcpy(A_device, A_host, N_data * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B_host, N_data * sizeof(float), hipMemcpyHostToDevice);

    auto mid3 = high_resolution_clock::now();

    unsigned long long int N_thread_per_block = 256;
    unsigned long long int N_block = (N_data - 0.5) / N_thread_per_block + 1;

    std::cout <<  " N_block: " << N_block <<  std::endl;
    std::cout <<  " N_thread_per_block: " << N_thread_per_block <<  std::endl;

    vec_add<<<N_block, N_thread_per_block>>>(A_device, B_device, C_device, N_data, N_ops);
    hipDeviceSynchronize();

    auto mid4 = high_resolution_clock::now();

    hipMemcpy(C_host, C_device, N_data * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Printing last 10 result" << std::endl;
    for (unsigned int i = N_data - 10; i < N_data; i++)
    {
        std::cout <<  " i: " << i <<  " C_host[i]: " << C_host[i] <<  std::endl;
    }

    // Free allocated memory
    free(A_host);
    free(B_host);
    free(C_host);
    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);

    auto end = high_resolution_clock::now();

    float time_init = duration_cast<microseconds>(mid1 - start).count() / 1000.;
    float time_allo = duration_cast<microseconds>(mid2 - mid1).count() / 1000.;
    float time_send = duration_cast<microseconds>(mid3 - mid2).count() / 1000.;
    float time_exec = duration_cast<microseconds>(mid4 - mid3).count() / 1000.;
    float time_retr = duration_cast<microseconds>(end - mid4).count() / 1000.;
    float time_tota = duration_cast<microseconds>(end - start).count() / 1000.;

    std::cout <<  " time_init: " << time_init <<  std::endl;
    std::cout <<  " time_allo: " << time_allo <<  std::endl;
    std::cout <<  " time_send: " << time_send <<  std::endl;
    std::cout <<  " time_exec: " << time_exec <<  std::endl;
    std::cout <<  " time_retr: " << time_retr <<  std::endl;
    std::cout <<  " time_tota: " << time_tota <<  std::endl;

    std::cout << "Result:," << time_init << "," << time_allo << "," << time_send << "," << time_exec << "," << time_retr << "," << time_tota << std::endl;

    return 0;
}
