
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define myInt_t unsigned long long

__global__ void setup_curandState(hiprandState* state)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}

__global__ void throw_dart(hiprandState* state, myInt_t* n_inside)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    double x = hiprand_uniform(&state[idx]);
    double y = hiprand_uniform(&state[idx]);
    double d = sqrt(x * x + y * y);
    if (d <= 1)
    {
        atomicAdd(n_inside, 1);
    }
}

int main()
{
    //~*~*~*~*~*~*~*~*~*~*~
    // Defining dimensions
    //~*~*~*~*~*~*~*~*~*~*~

    // we will launch 2048 blocks
    myInt_t grid_size = pow(2, 16);

    // we will generate 512 points each block
    myInt_t block_size = 512;

    // total threads
    myInt_t n_total_threads = grid_size * block_size;

    // create a pointer to the array of random state
    // each random state can be used to generate random number
    hiprandState* state_device;

    // malloc array of random state
    hipMalloc((void**) &state_device, n_total_threads * sizeof(hiprandState));

    // actually setup each random state with different index
    setup_curandState<<<grid_size, block_size>>>(state_device);

    // wait until all threads are done
    hipDeviceSynchronize();

    // setup a counter
    myInt_t* n_inside_device;

    // allocate memory
    hipMalloc((void**) &n_inside_device, sizeof(myInt_t));

    // actually throw the dart and count how many are inside
    throw_dart<<<grid_size, block_size>>>(state_device, n_inside_device);

    // wait until all threads are done
    hipDeviceSynchronize();

    // create a counter on host to copy device number to
    myInt_t* n_inside_host = new myInt_t;

    // copy the result to host
    hipMemcpy(n_inside_host, n_inside_device, sizeof(myInt_t), hipMemcpyDeviceToHost);

    // estimate pi by counting fraction
    double pi_estimate = (double) *n_inside_host / n_total_threads * 4.;

    // print pi_estimate
    std::cout <<  " pi_estimate: " << pi_estimate <<  std::endl;

    return 0;
}
