
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <random>
#include <chrono>
using namespace std::chrono;

void help()
{
    std::cout << "Usage:" << std::endl;
    std::cout << std::endl;
    std::cout << "    ./mycircle N_darts" << std::endl;
    std::cout << std::endl;
    std::cout << std::endl;
    return;
}


void count_darts_host(float* x, float* y, unsigned long long int* counter, int N_darts)
{
    for (unsigned i_task = 0; i_task < N_darts; ++i_task)
    {
        // compute the distance of the dart from the origin
        float dist = sqrt(x[i_task] * x[i_task] + y[i_task] * y[i_task]);

        // if the distance is less than 1 then count them as inside
        if (dist <= 1)
        {
            // atomic add
            *counter += 1;
        }
    }
}

int main(int argc, char** argv)
{

    if (argc < 3)
    {
        help();
        return 1;
    }

    auto start = high_resolution_clock::now();
    unsigned long long int N_darts = strtoull(argv[1], nullptr, 10);
    unsigned long long int N_thread_per_block = strtoull(argv[2], nullptr, 10);

    //~*~*~*~*~*~*~*~*~*~*~*~*~
    // Random Number Generator
    //~*~*~*~*~*~*~*~*~*~*~*~*~

    // create a random device
    std::random_device rd;

    // create a mersenne twistor rng seeded with the random device
    std::mt19937 gen(rd());

    // create a uniform real distribution between [0.0, 1.0)
    std::uniform_real_distribution<> distr(0.0, 1.0);


    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~
    // Create a list of random (x, y)
    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~

    // create a host (x, y) positions
    float* x_host = new float[N_darts];
    float* y_host = new float[N_darts];

    // Generate a random (x, y) positions
    for (unsigned int i = 0; i < N_darts; ++i)
    {
        x_host[i] = distr(gen);
        y_host[i] = distr(gen);
    }

    // create a counter_host
    unsigned long long int* counter_host = new unsigned long long int;

    auto mid = high_resolution_clock::now();

    count_darts_host(x_host, y_host, counter_host, N_darts);

    double pi_estimate = ((double)*counter_host) / N_darts * 4.;

    std::cout <<  " pi_estimate: " << pi_estimate <<  std::endl;

    auto stop = high_resolution_clock::now();

    auto duration_1 = duration_cast<microseconds>(mid - start);
    auto duration_2 = duration_cast<microseconds>(stop - mid);

    std::cout <<  " duration_1.count(): " << duration_1.count() <<  std::endl;
    std::cout <<  " duration_2.count(): " << duration_2.count() <<  std::endl;

    return 0;

}
