#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <random>
#include <chrono>
using namespace std::chrono;

//__________________________________________________________________________________________
void count_darts_host(float* x, float* y, unsigned long long int* counter, int N_darts)
{
    for (unsigned i_task = 0; i_task < N_darts; ++i_task)
    {
        // compute the distance of the dart from the origin
        float dist = sqrt(x[i_task] * x[i_task] + y[i_task] * y[i_task]);

        // if the distance is less than 1 then count them as inside
        if (dist <= 1)
        {
            // atomic add
            *counter += 1;
        }
    }
}





//__________________________________________________________________________________________
int main(int argc, char** argv)
{

    //~*~*~*~*~*~*~*~*~*~*~*~*~
    // Option settings
    //~*~*~*~*~*~*~*~*~*~*~*~*~

    unsigned long long N_darts = 1000000; // 1 million random points
    unsigned long long N_thread_per_block = 256; // 256 threads

    // If arguments are provided overwrite the default setting
    if (argc > 2)
    {
        N_darts = strtoull(argv[1], nullptr, 10);
        N_thread_per_block = strtoull(argv[2], nullptr, 10);
    }
    else if (argc > 1)
    {
        N_darts = strtoull(argv[1], nullptr, 10);
    }

    //~*~*~*~*~*~*~*~*~*~*~*~*~
    // Random Number Generator
    //~*~*~*~*~*~*~*~*~*~*~*~*~

    // create a random device
    std::random_device rd;

    // create a mersenne twistor rng seeded with the random device
    std::mt19937 gen(rd());

    // create a uniform real distribution between [0.0, 1.0)
    std::uniform_real_distribution<> distr(0.0, 1.0);


    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~
    // Create a list of random (x, y)
    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~

    // create a host (x, y) positions
    float* x_host = new float[N_darts];
    float* y_host = new float[N_darts];

    // Generate a random (x, y) positions
    for (unsigned int i = 0; i < N_darts; ++i)
    {
        x_host[i] = distr(gen);
        y_host[i] = distr(gen);
    }

    // create a counter_host
    unsigned long long int* counter_host = new unsigned long long int;

    auto mid = high_resolution_clock::now();

    count_darts_host(x_host, y_host, counter_host, N_darts);

    double pi_estimate = ((double)*counter_host) / N_darts * 4.;

    std::cout <<  " pi_estimate: " << pi_estimate <<  std::endl;

    auto stop = high_resolution_clock::now();

    auto duration_1 = duration_cast<microseconds>(mid - start);
    auto duration_2 = duration_cast<microseconds>(stop - mid);

    std::cout <<  " duration_1.count(): " << duration_1.count() <<  std::endl;
    std::cout <<  " duration_2.count(): " << duration_2.count() <<  std::endl;

    return 0;

}
