
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <random>
#include <chrono>
using namespace std::chrono;

//__________________________________________________________________________________________
void count_darts_host(float* x, float* y, unsigned long long int* counter, int N_darts)
{
    for (unsigned i_task = 0; i_task < N_darts; ++i_task)
    {
        // compute the distance of the dart from the origin
        float dist = sqrt(x[i_task] * x[i_task] + y[i_task] * y[i_task]);

        // if the distance is less than 1 then count them as inside
        if (dist <= 1)
        {
            // atomic add
            *counter += 1;
        }
    }
}





//__________________________________________________________________________________________
int main(int argc, char** argv)
{

    //~*~*~*~*~*~*~*~*~*~*~*~*~
    // Option settings
    //~*~*~*~*~*~*~*~*~*~*~*~*~

    int N_repeat = 1;
    unsigned long long N_darts = 1000000; // 1 million random points
    bool do_overlap_transfer = false;

    // If arguments are provided overwrite the default setting
    if (argc > 2)
    {
        N_repeat = atoi(argv[1]);
        N_darts = strtoull(argv[2], nullptr, 10);
    }
    else if (argc > 1)
    {
        N_repeat = atoi(argv[1]);
    }

    //~*~*~*~*~*~*~*~*~*~*~*~*~
    // Random Number Generator
    //~*~*~*~*~*~*~*~*~*~*~*~*~

    // create a random device
    std::random_device rd;

    // create a mersenne twistor rng seeded with the random device
    std::mt19937 gen(rd());

    // create a uniform real distribution between [0.0, 1.0)
    std::uniform_real_distribution<> distr(0.0, 1.0);

    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~
    // The "Answer"
    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~
    // Create a counter_dart_inside
    // This will count total number of how many fell
    // inside the quarter-circle in all tries
    // Then once we count how many total inside vs. total thrown,
    // from there we can estimate the pi by taking the fraction
    // Since the circle is a unit circle the area is supposed to be pi/4.
    // So the fraction should equal to pi/4.
    unsigned long long counter_dart_inside = 0;


    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~
    // Create a list of random (x, y)
    //~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~*~

    for (int i = 0; i < N_repeat; ++i)
    {

        // create a host (x, y) positions
        float* x_host = new float[N_darts];
        float* y_host = new float[N_darts];

        // Generate a random (x, y) positions
        for (unsigned int i = 0; i < N_darts; ++i)
        {
            x_host[i] = distr(gen);
            y_host[i] = distr(gen);
        }

        // create a counter_host
        unsigned long long int* counter_host = new unsigned long long int;

        count_darts_host(x_host, y_host, counter_host, N_darts);

        // Add to the grand counter
        counter_dart_inside += *counter_host;

    }

    double pi_estimate = ((double)counter_dart_inside) / (N_darts * N_repeat) * 4.;

    std::cout <<  " pi_estimate: " << pi_estimate <<  std::endl;

    return 0;

}
